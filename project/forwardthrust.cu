#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

#define threadsPerBlock 1024
#define numberOfBlocks 46
#define FILTER -1

// ptr =  cuda device pointer
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}

__global__ void nodeArray(int* dev_edges, int *dev_nodes,int size, int n){


    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int start = 0;
    int end = 0;
     
    for(int edgeIndex = idx*2 + 1; (edgeIndex + 2) < size ; edgeIndex += step){

        //use to calculate the degree of the last node
        if(edgeIndex == 1){
            // dev_nodes[n] = size >> 1;
            dev_nodes[0] = 0;
        }    
    
        int x = dev_edges[edgeIndex];
        // early stopping condition or
        // outofbound condition
         
        int y = dev_edges[edgeIndex + 2];
        
        if(x != y){

            start = x;
            end   = y;
        }

        else if (x == y && edgeIndex + 2 == size-1){

            start = x;
            end = n;
            edgeIndex += 2; 
            // printf("condition = %d %d\n",start,end);
        }

        // dealing with missing nodes
        for(int i = start+1 ; i <= end ; i++ ){
            dev_nodes[i] = (edgeIndex  + 2) >> 1; //always divisble by two
        }

    }
}    

__global__ void filter(int* dev_edges,int* dev_nodes,int size){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for(id = id*2; id < size ; id += step){

        int source = dev_edges[id];
        int destination   = dev_edges[id+1];

        int sourceDegree = dev_nodes[source+1] - dev_nodes[source];
        int destinationDegree = dev_nodes[destination+1] - dev_nodes[destination]; 


        if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && destination < source)){
            dev_edges[id] = FILTER;
            dev_edges[id + 1] = FILTER;
        }         
    }
}    

__global__ void trianglecounting(int* dev_edges,int* dev_nodes, int* result, int numberOfEdges){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;


    for( id = id * 2; id<numberOfEdges ; id = id+step){
    
        int count = 0;
        int s = dev_edges[id];
        int e = dev_edges[id + 1];

        int s_start = dev_nodes[s];
        int s_end = dev_nodes[s + 1];
        int e_start = dev_nodes[e];
        int e_end = dev_nodes[e + 1];
    
        int s_next,e_next;
        while(s_start < s_end && e_start < e_end)
        {
            s_next = dev_edges[(s_start << 1) + 1];
            e_next = dev_edges[(e_start << 1) + 1];
            int difference = s_next - e_next;
            if(difference == 0)
                count++;
            if(difference <= 0)
                s_start += 1;
            if(difference >= 0)
                e_start += 1;
        }

    result[id >> 1] = count;

    }

}    

void remove(int* dev_edges,int numberOfEdges){

    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , -1);

}

void sort(int* dev_edges,int numberOfEdges){

    // sort the edges
    thrust::device_ptr<uint64_t> ptr((uint64_t*)dev_edges);
    thrust::sort(ptr, ptr + numberOfEdges);
}

void parallelForward(const Edges& edges){

    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int* result;
    int numberOfNodes;

    
    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfEdges * sizeof(int));
    
    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);


    // debug(dev_edges,2*numberOfEdges,"orig data");
    sort(dev_edges,numberOfEdges);
    debug(dev_edges,2*numberOfEdges,"sort data");

    // Hardcoding the node value 
    // numberOfNodes = 1696415;
    numberOfNodes = 4;
     
    // allocate space for the node array
    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));

    // reuse the same node-array for everything to save space
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2,numberOfNodes);
    hipDeviceSynchronize();

    debug(dev_nodes,numberOfNodes+1,"node array");
     
    // compute the degree of the nodes
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2);
    hipDeviceSynchronize();

    
    //remove the filtered edges
    remove(dev_edges,numberOfEdges);

    debug(dev_edges,numberOfEdges,"filtred array");

    //get the node array once again
    //note = new size of the edge array is now numberOfEdges
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipDeviceSynchronize(); 

    debug(dev_nodes,numberOfNodes+1,"node array");
    
    // note = the actual index of the element in edge array is 2*nodeArray[i]
    cout<<"start triangle counting"<<endl;
    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, dev_nodes, result, numberOfEdges);
    hipDeviceSynchronize();

    //calculate the number of triangles
    thrust::device_ptr<int> ptr(result);
    int numberoftriangles =  thrust::reduce(ptr, ptr + numberOfNodes);

    // debug(result,numberOfNodes,"triangle array");

    printf("number of triangles = %d\n",numberoftriangles);

    hipFree(dev_edges);
    hipFree(dev_nodes);
    hipFree(result);


}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}
