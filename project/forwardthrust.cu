#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include<iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

#define threadsPerBlock 1024
#define FILTER -1

// ptr =  cuda device pointer
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}

__global__ void nodeArray(int* dev_edges, int *dev_nodes,int size, int n){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int start = 0;
    int end = 0;
     
    int edgeIndex = id * 2;

    //use to calculate the degree of the last node
    if(edgeIndex == 0)
       dev_nodes[n] = size >> 1;
    
    int x = dev_edges[edgeIndex];
    // early stopping condition or
    // outofbound condition
    if(x == n-1 || (edgeIndex + 2) >= size)
      return;

    int y = dev_edges[edgeIndex + 2];
    if(x != y){

        start = x;
        end   = y;
    }

   for(int i = start+1 ; i <= end ; i++ ){
       dev_nodes[i] = (edgeIndex + 2) >> 1; //always divisble by two
   }


}

__global__ void filter(int* dev_edges,int* dev_nodes,int numberOfEdges){

    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // access every second element
    id = id << 1; 

    // outofbound access
    if(id >= 2*numberOfEdges)
       return;

    int source = dev_edges[id];
    int destination   = dev_edges[id+1];

    int sourceDegree = dev_nodes[source+1] - dev_nodes[source];
    int destinationDegree = dev_nodes[destination+1] - dev_nodes[destination]; 


    if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && destination < source)){
        dev_edges[id] = FILTER;
        dev_edges[id + 1] = FILTER;
    }         

}

__global__ void trianglecounting(int* dev_edges,int* dev_nodes, int* result, int numberOfEdges){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    id = id << 1;
    // TODO: need to decide how many edges thread will be responsible for
    if(id >= numberOfEdges)
        return;
    int count = 0;
    int s = dev_edges[id];
    int e = dev_edges[id + 1];

    int s_start = dev_nodes[s];
    int s_end = dev_nodes[s + 1];
    int e_start = dev_nodes[e];
    int e_end = dev_nodes[e + 1];
    
    int s_next,e_next;
    while(s_start < s_end && e_start < e_end)
    {
        s_next = dev_edges[(s_start << 1) + 1];
        e_next = dev_edges[(e_start << 1) + 1];
        int difference = s_next - e_next;
        if(difference == 0)
            count++;
        if(difference <= 0)
            s_start += 1;
        if(difference >= 0)
            e_start += 1;
    }

    result[id >> 1] = count;



}

void parallelForward(const Edges& edges){

    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int *result;
    int numberOfBlocks;
    int numberOfNodes;
    int newBound;

    // TODO-: sort the edges
    
    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfEdges * sizeof(int));
    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);

    // Hardcoding the node value 
    numberOfNodes = 7;
     
    // allocate space for the node array
    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));

    // reuse the same node-array for everything to save space
    numberOfBlocks = (numberOfEdges + threadsPerBlock - 1) / threadsPerBlock;
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2,numberOfNodes);
    hipDeviceSynchronize();

    debug(dev_nodes,numberOfNodes+1,"print node array");
    
    // compute the degree of the nodes
    numberOfBlocks = (numberOfEdges + threadsPerBlock - 1) / threadsPerBlock;
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges);
    hipDeviceSynchronize();

    debug(dev_edges,numberOfEdges*2,"print filtered Edges");


    //remove the filtered edges
    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , -1);
    hipDeviceSynchronize();

    printf("number of edges = %d\n",numberOfEdges);
    debug(dev_edges,numberOfEdges ,"print filtered Edges");

    //get the node array once again
    //note = new size of the edge array is now numberOfEdges
    numberOfBlocks = (numberOfEdges/2 + threadsPerBlock - 1) / threadsPerBlock;
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipDeviceSynchronize(); 
    // note = the actual index of the element in edge array is 2*nodeArray[i]

    debug(dev_nodes,numberOfNodes+1,"print new node array");

    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, dev_nodes, result, numberOfEdges);
    hipDeviceSynchronize();

    debug(result,numberOfEdges/2,"print result array");
    //calculate the number of triangles
//    trianglecounting();


}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}
