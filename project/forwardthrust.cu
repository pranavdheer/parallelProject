#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

#define threadsPerBlock 1024
#define numberOfBlocks 400
#define FILTER -2

// ptr =  cuda device pointer
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}

void compare(int *ptr1 , int *ptr2 ,int size){

    int* deb1 = (int*)malloc(size * sizeof(int));
    int* deb2 = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb1,ptr1, size * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(deb2,ptr2, size * sizeof(int), hipMemcpyDeviceToHost);


    for(int i=0 ;i<size;i++)
        if(deb1[i] != deb2[i])
          cout<<i<<" "<<deb1[i]<<" "<<deb2[i]<<endl;

    free(deb1);
    free(deb2);

}

__global__ void nodeArray(int* dev_edges, int *dev_nodes,int size, int n){


    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int x,y;

    // bug-: node id that were not present, were not getting updated to zero 
    // eg-: 0 and 1 is not present they should have nodeattay index as 0 [resolved]

    if (idx == 0){

        x = dev_edges[1];
        for(int i=0 ;i<=x;i++)
            dev_nodes[i] = 0;

    }
    
    for( int id = idx; id < size/2; id += step){
        
        int edgeIndex = (id * 2) + 1;
        
        x = dev_edges[edgeIndex];
        
        if(id == size/2 - 1)
          y = n;
        
        else  
          y = dev_edges[edgeIndex + 2];
        
        // dealing with missing nodes
        for(int i = x+1 ; i <= y ; i++ ){  
            dev_nodes[i] = id + 1; //always divisble by two
        }

    }
}    

__global__ void filter(int* dev_edges,int* dev_nodes,int size){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for(id = id*2; id < size ; id += step){

        int source = dev_edges[id];
        int destination   = dev_edges[id+1];

        int sourceDegree = dev_nodes[source+1] - dev_nodes[source];
        int destinationDegree = dev_nodes[destination+1] - dev_nodes[destination]; 


        if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && destination < source)){
            dev_edges[id] = FILTER;
            dev_edges[id + 1] = FILTER;
        }         
    }
}    

__global__ void trianglecounting(int* dev_edges,int* dev_nodes, int* result, int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;    
    int count = 0;
    int id = 0;
    
    for(int iter = idx; iter < numberOfEdges/2; iter = iter + step)
    {
      
        id = iter *2;
        int s = dev_edges[id];
        int e = dev_edges[id + 1];

        int s_start = dev_nodes[s];
        int s_end = dev_nodes[s + 1];
        int e_start = dev_nodes[e];
        int e_end = dev_nodes[e + 1];
        int s_next,e_next;

        // printf("id = %d,s = %d, s_start = %d \n",id,s,s_start);
        while(s_start < s_end && e_start < e_end)
        {
            s_next = dev_edges[(s_start << 1)];
            e_next = dev_edges[(e_start << 1)];
            int difference = s_next - e_next;
            // printf("I am here %d\n",difference);
            if(difference == 0)
                count++;
            if(difference <= 0)
                s_start += 1;
            if(difference >= 0)
                e_start += 1;
        }
    }
    result[idx] = count;

}    

void remove(int* dev_edges,int numberOfEdges){

    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , FILTER);

}

void sort(int* dev_edges,int numberOfEdges){

    // sort the edges
    thrust::device_ptr<uint64_t> ptr((uint64_t*)dev_edges);
    thrust::sort(ptr, ptr + numberOfEdges);
}

void parallelForward(const Edges& edges){

    double startKernelTime = CycleTimer::currentSeconds();

    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int* result;
    int numberOfNodes;

    hipEvent_t startNodeArray1, stopNodeArray1, startNodeArray2, stopNodeArray2,startFilter, 
    stopFilter, startTriCount, stopTriCount;

    // timer code
    hipEventCreate(&startNodeArray1);
    hipEventCreate(&stopNodeArray1);

    hipEventCreate(&startNodeArray2);
    hipEventCreate(&stopNodeArray2);

    hipEventCreate(&startFilter);
    hipEventCreate(&stopFilter);

    hipEventCreate(&startTriCount);
    hipEventCreate(&stopTriCount);
     

    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfBlocks * threadsPerBlock * sizeof(int));


    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);

    sort(dev_edges,numberOfEdges);


    // Hardcoding the node value 
    numberOfNodes = 1696415;
    // numberOfNodes = 6;

    // allocate space for the node array
    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));

    hipEventRecord(startNodeArray1);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2,numberOfNodes);
    hipEventRecord(stopNodeArray1);

    hipDeviceSynchronize();

     
    // compute the degree of the nodes
    hipEventRecord(startFilter);
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2);
    hipEventRecord(stopFilter);

    hipDeviceSynchronize();

    //remove the filtered edges
    remove(dev_edges,numberOfEdges);


    //get the node array once again
    //note = new size of the edge array is now numberOfEdges

    hipEventRecord(startNodeArray2);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipEventRecord(stopNodeArray2);

    hipDeviceSynchronize(); 

    hipEventRecord(startTriCount);
    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, dev_nodes, result, numberOfEdges);
    hipEventRecord(stopTriCount);

    hipDeviceSynchronize();

    //calculate the number of triangles
    thrust::device_ptr<int> ptr(result);
    int numberoftriangles =  thrust::reduce(ptr, ptr + (numberOfBlocks * threadsPerBlock));

    // debug(result,numberOfNodes,"triangle array");

    printf("number of triangles = %d\n",numberoftriangles);

    hipFree(dev_edges);
    hipFree(dev_nodes);
    hipFree(result);

    float m1 = 0;
    hipEventElapsedTime(&m1, startNodeArray1, stopNodeArray1);
    printf("CUDA Elapsed Time for Node Array 1 = %f ms\n", m1);

    float m2 = 0;
    hipEventElapsedTime(&m2, startFilter, stopFilter);
    printf("CUDA Elapsed Time for Filter = %f ms\n", m2);

    float m3 = 0;
    hipEventElapsedTime(&m3, startNodeArray2, stopNodeArray2);
    printf("CUDA Elapsed Time for Node Array 2 = %f ms\n", m3);

    float m4 = 0;
    hipEventElapsedTime(&m4, startTriCount, stopTriCount);
    printf("CUDA Elapsed Time for Triangle Counting = %f ms\n", m4);

    hipEventDestroy(startNodeArray1);
    hipEventDestroy(stopNodeArray1);
    hipEventDestroy(startNodeArray2);
    hipEventDestroy(stopNodeArray2);
    hipEventDestroy(startFilter);
    hipEventDestroy(stopFilter);
    hipEventDestroy(startTriCount);
    hipEventDestroy(stopTriCount);
    
    
    double endKernelTime = CycleTimer::currentSeconds();
    double kernelDuration = endKernelTime - startKernelTime;
    printf("KernelDuration: %.3f ms\n", 1000.f * kernelDuration);


}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}
