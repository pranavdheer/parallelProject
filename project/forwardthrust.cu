#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include<iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

#define threadsPerBlock 1024
#define numberOfBlocks 46
#define FILTER -1
#define inf 0x7f800000 
// ptr =  cuda device pointer
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}


__device__ void warp_reduce_max(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] > smem[threadIdx.x] ? 
                        smem[threadIdx.x+512] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+256] > smem[threadIdx.x] ? 
						smem[threadIdx.x+256] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+128] > smem[threadIdx.x] ? 
						smem[threadIdx.x+128] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+64] > smem[threadIdx.x] ? 
						smem[threadIdx.x+64] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+32] > smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+16] > smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+8] > smem[threadIdx.x] ? 
						smem[threadIdx.x+8] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+4] > smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+2] > smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+1] > smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; __syncthreads();

}
__global__ void find_max_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_max[1024];

	int idx = threadIdx.x + remaining;

	int max = -inf;
	int val;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		val = in[idx + iter];
		max = val > max ? val : max;
        iter = i * threadsPerBlock;
    }

	iter = 0;
	for(int i = 0; (iter + threadIdx.x) < num_blocks; i++)
	{
		val = out[threadIdx.x + iter];
		max = val > max ? val : max;
		iter = i * threadsPerBlock;
	}

	smem_max[threadIdx.x] = max;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
}

__global__ void find_max(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_max[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;

	int max = -inf;

	int val;

	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
    {
        val = in[idx + i * threadsPerBlock];
        max = val > max ? val : max;

    }

	smem_max[threadIdx.x] = max;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);
	

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
	

}

void calculateNumVertices(int* d_in, int* d_out, int num_elements)
{

	//int elements_per_block = ; // needs to be set (random right now) ( = m * 2 / number of blocks)
		
	int num_blocks = numberOfBlocks;//46;//num_elements / elements_per_block; // redundant
    int elements_per_block = num_elements / num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;
    //printf("helo\n");
	find_max<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_max_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);
    //printf("dout %d \n", *d_out );
	
}

__global__ void nodeArray(const int* __restrict__ dev_edges, int *dev_nodes,int size, int n){
/*
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int start = 0;
    int end = 0;
     
    for(int id = idx; ((id * 2) + 2) < size ; id += step){

        int edgeIndex = (id * 2) + 1;        
        //use to calculate the degree of the last node
        if(edgeIndex == 1){
            // dev_nodes[n] = size >> 1;
            dev_nodes[0] = 0;
        }    
    
        int x = dev_edges[edgeIndex];
        // early stopping condition or
        // outofbound condition
         
        int y = dev_edges[edgeIndex + 2];
        
        if(x != y){

            start = x;
            end   = y;
        }

        else if (x == y && edgeIndex + 2 == size-1){

            start = x;
            end = n;
            edgeIndex += 2; 
            // printf("condition = %d %d\n",start,end);
        }

        // dealing with missing nodes
        for(int i = start+1 ; i <= end ; i++ ){
            dev_nodes[i] = (edgeIndex  + 2) >> 1; //always divisble by two
        }

    }
    */
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = numberOfBlocks * threadsPerBlock;
    int prev, next;

    for (int i = idx; i <= size; i += step) 
    {
        // to remove the extra predication, all threads execute else
        prev = -1;
        if(i > 0)
            prev = dev_edges[(2 * (i - 1)) + 1];
        //else 
        //    prev = -1;
        next = n;
        if(i < size)
            next = dev_edges[(2 * i) + 1];
        //else
        //    next = n;
        //int prev = i > 0 ? dev_edges[(2 * (i - 1) + 1)] : -1;
        //int next = i < size ? dev_edges[(2 * i + 1)] : n;
        for (int j = prev + 1; j <= next; ++j)
            dev_nodes[j] = i;
    }


}

__global__ void filter(int* dev_edges,const int* __restrict__ dev_nodes,int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    //int id;
    for(int iter = idx; iter < numberOfEdges; iter += step)
    {
        // access every second element
        //id = iter * 2; 


        int2 sd_pair = ((int2*)dev_edges)[iter];
        // can't use int2 because int2 pointer moves in multiples of 2
        int sourceDegree = dev_nodes[(sd_pair.x) + 1] - dev_nodes[sd_pair.x];
        int destinationDegree = dev_nodes[(sd_pair.y) + 1] - dev_nodes[sd_pair.y];
        /*
        int source = dev_edges[id];
        int destination   = dev_edges[id + 1];

        int sourceDegree = dev_nodes[source+1] - dev_nodes[source];
        int destinationDegree = dev_nodes[destination+1] - dev_nodes[destination]; 
        */

        //if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && destination < source)){
        if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && sd_pair.y < sd_pair.x)){
            //dev_edges[id] = FILTER;
            //dev_edges[id + 1] = FILTER;
            ((int2*)dev_edges)[iter] =  make_int2(FILTER, FILTER);
        }    
    }     

}

__global__ void trianglecounting(const int* __restrict__ dev_edges,const int* __restrict__ dev_nodes, uint64_t* result, int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int count  = 0;
    //int id = 0;
    for(int iter = idx; iter<numberOfEdges / 2; iter = iter+step){

        //id = iter * 2;
        
        /*
        int s = dev_edges[id];
        int e = dev_edges[id + 1];
        */
        int2 se_pair = ((int2*)dev_edges)[iter];
        int s_start = dev_nodes[se_pair.x];
        int s_end = dev_nodes[se_pair.x + 1];
        int e_start = dev_nodes[se_pair.y];
        int e_end = dev_nodes[se_pair.y + 1];
        
        int2 s_next,e_next;

        while(s_start < s_end && e_start < e_end)
        {
            /*
            s_next = dev_edges[(s_start << 1)];
            e_next = dev_edges[(e_start << 1)];
            int difference = s_next - e_next;
            if(difference < 0)
                s_start += 1;
            else if(difference > 0)
                e_start += 1;
            else {
                s_start += 1;
                e_start += 1;
                count++;
            }
            */
            // TODO: need to run and check for speed, vector accesses might have increased execution time
            s_next = ((int2*)dev_edges)[s_start];
            e_next = ((int2*)dev_edges)[e_start];

            if(s_next.x <= e_next.x)
                s_start+=1;
            if(s_next.x >= e_next.x)
                e_start+=1;
            if(s_next.x == e_next.x)
                count++;
            
        }

    

    }
    result[idx] = count;
    //dev_edges[numberOfEdges + idx] = count;
}
void SortEdges(int m, int* edges) {
    thrust::device_ptr<uint64_t> ptr((uint64_t*)edges);
    thrust::sort(ptr, ptr + m);
    }
void remove(int* dev_edges,int numberOfEdges){

    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , -1);

}
int NumVerticesGPU(int m, int* edges) {
    thrust::device_ptr<int> ptr(edges);
    return 1 + thrust::reduce(ptr, ptr + 2 * m, 0, thrust::maximum<int>());
    }
void parallelForward(const Edges& edges){

    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int* d_out;
    uint64_t *result;
    //int numberOfBlocks;
    int numberOfNodes;
    int* out = (int*)malloc(sizeof(int));
    // TODO-: sort the edges
    
    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfBlocks * threadsPerBlock * sizeof(uint64_t));
    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);
    hipMalloc(&d_out, 2 * numberOfEdges * sizeof(int));

    // Hardcoding the node value 
    calculateNumVertices(dev_edges, d_out, numberOfEdges * 2);
    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    numberOfNodes = 1 + (*out);
    printf("number of nodes = %d\n", numberOfNodes);
    // numberOfNodes = 4;
    SortEdges(numberOfEdges, dev_edges);
    // allocate space for the node array
    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));

    // reuse the same node-array for everything to save space
    //numberOfBlocks = (numberOfEdges + threadsPerBlock - 1) / threadsPerBlock;
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipDeviceSynchronize();

    printf("number of edges = %d\n", numberOfEdges);
    // compute the degree of the nodes
    //numberOfBlocks = (numberOfEdges + threadsPerBlock - 1) / threadsPerBlock;
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges);
    hipDeviceSynchronize();

    //remove the filtered edges
    remove(dev_edges,numberOfEdges);
    //printf("hello\n");
    //get the node array once again
    //note = new size of the edge array is now numberOfEdges
    //numberOfBlocks = (numberOfEdges/2 + threadsPerBlock - 1) / threadsPerBlock;
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges/2,numberOfNodes);
    hipDeviceSynchronize(); 
    // note = the actual index of the element in edge array is 2*nodeArray[i]

    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, dev_nodes, result, numberOfEdges);
    hipDeviceSynchronize();


    //calculate the number of triangles
    //change pointer to int in case of using dev_edges
    //ptr = dev_edges + numberOfEdges
    thrust::device_ptr<uint64_t> ptr(result);
    uint64_t numberoftriangles =  thrust::reduce(ptr, ptr + (numberOfBlocks * threadsPerBlock));

    //debug(result,numberOfNodes,"triangle array");

    printf("number of triangles = %lld\n",numberoftriangles);
    hipFree(result);
    hipFree(dev_edges);
    hipFree(dev_nodes);

}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}
