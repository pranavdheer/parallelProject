// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include "lib.h"


void parallelForward(const Edges& edges){

    int m = edges.size();
    
    // transfer data to GPU
    hipMalloc(&dev_edges, m * 2 * sizeof(int));

    hipMemcpy(dev_edges, edges.data(), m * 2 * sizeof(int),
    hipMemcpyHostToDevice);








}

void
printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}