#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include<iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

#define threadsPerBlock 1024
#define numberOfBlocks 46
#define blocks_x_threads (threadsPerBlock * numberOfBlocks)
#define FILTER -1
#define inf 0x7f800000 
// ptr =  cuda device pointer
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}

__device__ void warp_reduce_sum(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+256] + smem[threadIdx.x]; __syncthreads();
    smem[threadIdx.x] = smem[threadIdx.x+128] + smem[threadIdx.x]; __syncthreads();
    smem[threadIdx.x] = smem[threadIdx.x+64] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+32] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+16] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+8] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+4] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+2] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+1] + smem[threadIdx.x]; __syncthreads();
}
__global__ void find_sum_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_sum[1024];

	int idx = threadIdx.x + remaining;

	int sum = 0;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		sum += in[idx + iter];
        iter = i * threadsPerBlock;
    }
	iter = 0;
	for(int i = 1; (iter + threadIdx.x) < num_blocks; i++)
	{
		sum += out[threadIdx.x + iter];
		iter = i * threadsPerBlock;
	}
	smem_sum[threadIdx.x] = sum;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_sum(smem_sum);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_sum[threadIdx.x]; 
}

__global__ void find_sum(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_sum[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;
	int sum = 0;
	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
        sum += in[idx + i * threadsPerBlock];

	smem_sum[threadIdx.x] = sum;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_sum(smem_sum);

	if(threadIdx.x == 0) 
		out[blockIdx.x] = smem_sum[threadIdx.x]; 
}

void calculateSum(int* d_in, int* d_out, int num_elements)
{
		
	int num_blocks = 46;
    int elements_per_block = num_elements/num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;

	find_sum<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_sum_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);
}


__device__ void warp_reduce_max(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] > smem[threadIdx.x] ? 
                        smem[threadIdx.x+512] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+256] > smem[threadIdx.x] ? 
						smem[threadIdx.x+256] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+128] > smem[threadIdx.x] ? 
						smem[threadIdx.x+128] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+64] > smem[threadIdx.x] ? 
						smem[threadIdx.x+64] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+32] > smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+16] > smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+8] > smem[threadIdx.x] ? 
						smem[threadIdx.x+8] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+4] > smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+2] > smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+1] > smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; __syncthreads();

}
__global__ void find_max_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_max[1024];

	int idx = threadIdx.x + remaining;

	int max = -inf;
	int val;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		val = in[idx + iter];
		max = val > max ? val : max;
        iter = i * threadsPerBlock;
    }

	iter = 0;
	for(int i = 1; (iter + threadIdx.x) < num_blocks; i++)
	{
		val = out[threadIdx.x + iter];
		max = val > max ? val : max;
		iter = i * threadsPerBlock;
	}

	smem_max[threadIdx.x] = max;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
}

__global__ void find_max(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_max[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;

	int max = -inf;

	int val;

	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
    {
        val = in[idx + i * threadsPerBlock];
        max = val > max ? val : max;

    }

	smem_max[threadIdx.x] = max;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);
	

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
	

}

void calculateNumVertices(int* d_in, int* d_out, int num_elements)
{
		
	int num_blocks = numberOfBlocks;
    int elements_per_block = num_elements / num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;
    //printf("helo\n");
	find_max<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_max_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);
    //printf("dout %d \n", *d_out );
	
}

__global__ void nodeArray(const int* __restrict__ dev_edges, int *dev_nodes,int size, int n){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = blocks_x_threads;
    int x,y;



    if (idx == 0){

        x = dev_edges[1];
        for(int i=0 ;i<=x;i++)
            dev_nodes[i] = 0;

    }
    
    for( int id = idx; id < size/2; id += step){
        
        int edgeIndex = (id * 2) + 1;
        
        x = dev_edges[edgeIndex];
        
        if(id == size/2 - 1)
          y = n;
        
        else  
          y = dev_edges[edgeIndex + 2];
        
        // dealing with missing nodes
        for(int i = x+1 ; i <= y ; i++ ){  
            dev_nodes[i] = id + 1; //always divisble by two
        }

    }
 


}

__global__ void filter(int* dev_edges,const int* __restrict__ dev_nodes,int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = blocks_x_threads;
    //int id;
    for(int iter = idx; iter < numberOfEdges; iter += step)
    {



        int2 sd_pair = ((int2*)dev_edges)[iter];
        // can't use int2 because int2 pointer moves in multiples of 2
        int sourceDegree = dev_nodes[(sd_pair.x) + 1] - dev_nodes[sd_pair.x];
        int destinationDegree = dev_nodes[(sd_pair.y) + 1] - dev_nodes[sd_pair.y];

        if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && sd_pair.y < sd_pair.x)){

            ((int2*)dev_edges)[iter] =  make_int2(FILTER, FILTER);
        }    
    }     

}
__global__ void separate(int* dev_edges, int*d_out, int numberOfEdges) {
    int from = blockDim.x * blockIdx.x + threadIdx.x;
    int step = blocks_x_threads;
    for (int i = from; i < numberOfEdges; i += step) {
        int2 sd_pair = ((int2*)dev_edges)[i];  
        d_out[i] = sd_pair.x;
        d_out[numberOfEdges + i] = sd_pair.y;

    }
}
__global__ void trianglecounting(const int* __restrict__ dev_edges,const int* __restrict__ dev_nodes, int* result, int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;    
    int count = 0;
    int id = 0;
    
    for(int iter = idx; iter < numberOfEdges; iter = iter + step)
    {
        
        id = iter;
        int s = dev_edges[id];
        int e = dev_edges[numberOfEdges + id];

        int s_start = dev_nodes[s];
        int s_end = dev_nodes[s + 1];
        int e_start = dev_nodes[e];
        int e_end = dev_nodes[e + 1];
        int s_next,e_next;
        s_next = dev_edges[(s_start)];
        e_next = dev_edges[(e_start)];


        while(s_start < s_end && e_start < e_end)
        {

            int difference = s_next - e_next;


            if(difference < 0) { 
                s_start += 1;
                s_next = dev_edges[(s_start)];
            }
            if(difference > 0) {
                e_start += 1;
                e_next = dev_edges[(e_start)];
            }
            if(difference == 0) {
                count++;
                s_start += 1;
                s_next = dev_edges[(s_start)];
                e_start += 1;
                e_next = dev_edges[(e_start)];
            }
        }
    }
    result[idx] = count;
}
void SortEdges(int m, int* edges) {
    thrust::device_ptr<uint64_t> ptr((uint64_t*)edges);
    thrust::sort(ptr, ptr + m);
    }
void remove(int* dev_edges,int numberOfEdges){

    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , -1);

}
int NumVerticesGPU(int m, int* edges) {
    thrust::device_ptr<int> ptr(edges);
    return 1 + thrust::reduce(ptr, ptr + 2 * m, 0, thrust::maximum<int>());
    }
void parallelForward(const Edges& edges){

    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int* d_out;
    int *result;
    //int numberOfBlocks;
    int numberOfNodes;
    int* out = (int*)malloc(sizeof(int));


    hipEvent_t startNodeArray1, stopNodeArray1, startNodeArray2, stopNodeArray2,startFilter, stopFilter, startTriCount, stopTriCount, startNumVertices, stopNumVertices, startSumTri, stopSumTri, startSeparate, stopSeparate;

    hipEventCreate(&startNodeArray1);
    hipEventCreate(&stopNodeArray1);
    hipEventCreate(&startNodeArray2);
    hipEventCreate(&stopNodeArray2);
    hipEventCreate(&startFilter);
    hipEventCreate(&stopFilter);
    hipEventCreate(&startTriCount);
    hipEventCreate(&stopTriCount);
    hipEventCreate(&startNumVertices);
    hipEventCreate(&stopNumVertices);
    hipEventCreate(&startSumTri);
    hipEventCreate(&stopSumTri);
    hipEventCreate(&startSeparate);
    hipEventCreate(&stopSeparate);

    double startKernelTime = CycleTimer::currentSeconds();
    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfBlocks * threadsPerBlock * sizeof(uint64_t));
    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);
    hipMalloc(&d_out, 2 * numberOfEdges * sizeof(int));




    // Hardcoding the node value 
    hipEventRecord(startNumVertices);
    calculateNumVertices(dev_edges, d_out, numberOfEdges * 2);
    hipEventRecord(stopNumVertices);
    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    numberOfNodes = 1 + (*out);
    printf("number of nodes = %d\n", numberOfNodes);

    SortEdges(numberOfEdges, dev_edges);
    // allocate space for the node array
    //debug(dev_edges,numberOfEdges*2,"sorted edges");

    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));


    hipEventRecord(startNodeArray1);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2,numberOfNodes);
    hipEventRecord(stopNodeArray1);
    hipDeviceSynchronize();
    //debug(dev_nodes,numberOfNodes,"node aray 1");

    printf("number of edges = %d\n", numberOfEdges);
    // compute the degree of the nodes

    hipEventRecord(startFilter);
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges);
    hipEventRecord(stopFilter);
    hipDeviceSynchronize();

    //remove the filtered edges
    remove(dev_edges,numberOfEdges);
    //debug(dev_edges,numberOfEdges,"filtered edges");

    hipEventRecord(startNodeArray2);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipEventRecord(stopNodeArray2);
    hipDeviceSynchronize(); 
    //debug(dev_nodes,numberOfNodes,"node aray 2");
    hipEventRecord(startSeparate);
    separate<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, d_out, numberOfEdges/2);
    hipEventRecord(stopSeparate);
    //debug(d_out,numberOfEdges,"separarted array");

    hipEventRecord(startTriCount);
    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(d_out, dev_nodes, result, numberOfEdges/2);
    hipEventRecord(stopTriCount);
    hipDeviceSynchronize();

    hipEventRecord(startSumTri);
    calculateSum(result, dev_edges, numberOfBlocks * threadsPerBlock);
    hipEventRecord(stopSumTri);



    //calculate the number of triangles

    hipMemcpy(out, dev_edges, sizeof(int), hipMemcpyDeviceToHost);
    int numberoftriangles = (*out);


    hipFree(result);
    hipFree(dev_edges);
    hipFree(dev_nodes);
    hipFree(d_out);

    
    double endKernelTime = CycleTimer::currentSeconds();
    //debug(result,numberOfNodes,"triangle array");
    free(out);
    printf("number of triangles = %lld\n",numberoftriangles);

    float m1 = 0;
    hipEventElapsedTime(&m1, startNodeArray1, stopNodeArray1);
    printf("CUDA Elapsed Time for Node Array 1 %f ms\n", m1);

    float m2 = 0;
    hipEventElapsedTime(&m2, startFilter, stopFilter);
    printf("CUDA Elapsed Time for Filter %f ms\n", m2);

    float m5 = 0;
    hipEventElapsedTime(&m5, startNumVertices, stopNumVertices);
    printf("CUDA Elapsed Time for Num Vertices %f ms\n", m5);

    float m3 = 0;
    hipEventElapsedTime(&m3, startNodeArray2, stopNodeArray2);
    printf("CUDA Elapsed Time for Node Array 2 %f ms\n", m3);

    float m7 = 0;
    hipEventElapsedTime(&m7, startSeparate, stopSeparate);
    printf("CUDA Elapsed Time for separate %f ms\n", m7);

    float m4 = 0;
    hipEventElapsedTime(&m4, startTriCount, stopTriCount);
    printf("CUDA Elapsed Time for Triangle Counting %f ms\n", m4);

    float m6 = 0;
    hipEventElapsedTime(&m6, startSumTri, stopSumTri);
    printf("CUDA Elapsed Time for Sum Triangles %f ms\n", m6);

    hipEventDestroy(startNodeArray1);
    hipEventDestroy(stopNodeArray1);
    hipEventDestroy(startNodeArray2);
    hipEventDestroy(stopNodeArray2);
    hipEventDestroy(startFilter);
    hipEventDestroy(stopFilter);
    hipEventDestroy(startTriCount);
    hipEventDestroy(stopTriCount);
    hipEventDestroy(startNumVertices);
    hipEventDestroy(stopNumVertices);
    hipEventDestroy(startSumTri);
    hipEventDestroy(stopSumTri);
    hipEventDestroy(startSeparate);
    hipEventDestroy(stopSeparate);


    double kernelDuration = endKernelTime - startKernelTime;
    printf("KernelDuration: %.3f ms\n", 1000.f * kernelDuration);

}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}
