#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

#define threadsPerBlock 1024
#define numberOfBlocks 400
#define FILTER -2
#define inf 0x7f800000 

// ptr =  cuda device pointer
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}

void compare(int *ptr1 , int *ptr2 ,int size){

    int* deb1 = (int*)malloc(size * sizeof(int));
    int* deb2 = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb1,ptr1, size * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(deb2,ptr2, size * sizeof(int), hipMemcpyDeviceToHost);


    for(int i=0 ;i<size;i++)
        if(deb1[i] != deb2[i])
          cout<<i<<" "<<deb1[i]<<" "<<deb2[i]<<endl;

    free(deb1);
    free(deb2);

}

__global__ void nodeArray(int* dev_edges, int *dev_nodes,int size, int n){


    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int x,y;

    // bug-: node id that were not present, were not getting updated to zero 
    // eg-: 0 and 1 is not present they should have nodeattay index as 0 [resolved]

    if (idx == 0){

        x = dev_edges[1];
        for(int i=0 ;i<=x;i++)
            dev_nodes[i] = 0;

    }
    
    for( int id = idx; id < size/2; id += step){
        
        int edgeIndex = (id * 2) + 1;
        
        x = dev_edges[edgeIndex];
        
        if(id == size/2 - 1)
          y = n;
        
        else  
          y = dev_edges[edgeIndex + 2];
        
        // dealing with missing nodes
        for(int i = x+1 ; i <= y ; i++ ){  
            dev_nodes[i] = id + 1; //always divisble by two
        }

    }
}    

__global__ void filter(int* dev_edges,int* dev_nodes,int numberOfEdges){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for(id = id; id < numberOfEdges ; id += step){

        int2 sd_pair = ((int2*)dev_edges)[id];

        int sourceDegree = dev_nodes[(sd_pair.x)+1] - dev_nodes[sd_pair.x];
        int destinationDegree = dev_nodes[(sd_pair.y) + 1] - dev_nodes[sd_pair.y]; 

        if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && sd_pair.y < sd_pair.x)){

            ((int2*)dev_edges)[id] =  make_int2(FILTER, FILTER);

        }         
    }
}   

__device__ void warp_reduce_max(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] > smem[threadIdx.x] ? 
                        smem[threadIdx.x+512] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+256] > smem[threadIdx.x] ? 
						smem[threadIdx.x+256] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+128] > smem[threadIdx.x] ? 
						smem[threadIdx.x+128] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+64] > smem[threadIdx.x] ? 
						smem[threadIdx.x+64] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+32] > smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+16] > smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+8] > smem[threadIdx.x] ? 
						smem[threadIdx.x+8] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+4] > smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+2] > smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+1] > smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; __syncthreads();
}

__global__ void find_max(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_max[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;
	int max = -inf;
	int val;
	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
    {
        val = in[idx + i * threadsPerBlock];
        max = val > max ? val : max;

    }

	smem_max[threadIdx.x] = max;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);
	
	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
	
}

__global__ void find_max_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_max[1024];

	int idx = threadIdx.x + remaining;

	int max = -inf;
	int val;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		val = in[idx + iter];
		max = val > max ? val : max;
        iter = i * threadsPerBlock;
    }

	iter = 0;
	for(int i = 1; (iter + threadIdx.x) < num_blocks; i++)
	{
		val = out[threadIdx.x + iter];
		max = val > max ? val : max;
		iter = i * threadsPerBlock;
	}

	smem_max[threadIdx.x] = max;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
}


void calculateNumVertices(int* d_in, int* d_out, int num_elements)
{

	//int elements_per_block = ; // needs to be set (random right now) ( = m * 2 / number of blocks)
		
	int num_blocks = numberOfBlocks;//46;//num_elements / elements_per_block; // redundant
    int elements_per_block = num_elements / num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;

	find_max<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_max_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);

	
}


__global__ void trianglecounting(const int* __restrict__ dev_edges,const int* __restrict__ dev_nodes, int* result, int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int count  = 0;
 
    for(int iter = idx; iter<numberOfEdges / 2; iter = iter+step){

        int2 se_pair = ((int2*)dev_edges)[iter];
        int s_start = dev_nodes[se_pair.x];
        int s_end = dev_nodes[se_pair.x + 1];
        int e_start = dev_nodes[se_pair.y];
        int e_end = dev_nodes[se_pair.y + 1];
        
        int2 s_next,e_next;
        s_next = ((int2*)dev_edges)[s_start];
        e_next = ((int2*)dev_edges)[e_start];

        while(s_start < s_end && e_start < e_end)
        {
            // need to run and check for speed, vector accesses might have increased execution time
            int a = s_next.x;
            int b = e_next.x;

            if(a < b) {
                s_start+=1;
                s_next = ((int2*)dev_edges)[s_start];
            }
            else if(a > b) {
                e_start+=1;
                e_next = ((int2*)dev_edges)[e_start];
            }
            else {
                count++;
                s_start+=1;
                s_next = ((int2*)dev_edges)[s_start];
                e_start+=1;
                e_next = ((int2*)dev_edges)[e_start];
            }   
        }
    }
    result[idx] = count;
}

void remove(int* dev_edges,int numberOfEdges){

    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , FILTER);

}

void sort(int* dev_edges,int numberOfEdges){

    // sort the edges
    thrust::device_ptr<uint64_t> ptr((uint64_t*)dev_edges);
    thrust::sort(ptr, ptr + numberOfEdges);
}

int NumVerticesGPU(int m, int* edges) {
    thrust::device_ptr<int> ptr(edges);
    return 1 + thrust::reduce(ptr, ptr + 2 * m, 0, thrust::maximum<int>());
}

__device__ void warp_reduce_sum(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+256] + smem[threadIdx.x]; __syncthreads();
    smem[threadIdx.x] = smem[threadIdx.x+128] + smem[threadIdx.x]; __syncthreads();
    smem[threadIdx.x] = smem[threadIdx.x+64] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+32] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+16] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+8] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+4] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+2] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+1] + smem[threadIdx.x]; __syncthreads();
}

__global__ void find_sum_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_sum[1024];

	int idx = threadIdx.x + remaining;

	int sum = 0;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		sum += in[idx + iter];
        iter = i * threadsPerBlock;
    }
	iter = 0;
	for(int i = 1; (iter + threadIdx.x) < num_blocks; i++)
	{
		sum += out[threadIdx.x + iter];
		iter = i * threadsPerBlock;
	}
	smem_sum[threadIdx.x] = sum;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_sum(smem_sum);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_sum[threadIdx.x]; 
}

__global__ void find_sum(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_sum[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;
	int sum = 0;
	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
        sum += in[idx + i * threadsPerBlock];

	smem_sum[threadIdx.x] = sum;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_sum(smem_sum);

	if(threadIdx.x == 0) 
		out[blockIdx.x] = smem_sum[threadIdx.x]; 
}

void calculateSum(int* d_in, int* d_out, int num_elements)
{
		
	int num_blocks = numberOfBlocks;
    int elements_per_block = num_elements/num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;

	find_sum<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_sum_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);
}

void parallelForward(const Edges& edges){

    double startKernelTime = CycleTimer::currentSeconds();

    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int* result;
    int* d_out;
    int numberOfNodes;
    int* out = (int*)malloc(sizeof(int));

    hipEvent_t startNodeArray1, stopNodeArray1, startNodeArray2, stopNodeArray2,startFilter, 
    stopFilter, startTriCount, stopTriCount,startNumvertices,stopNumvertices, startSumTri, stopSumTri;

    // timer code
    hipEventCreate(&startNodeArray1);
    hipEventCreate(&stopNodeArray1);

    hipEventCreate(&startNodeArray2);
    hipEventCreate(&stopNodeArray2);

    hipEventCreate(&startFilter);
    hipEventCreate(&stopFilter);

    hipEventCreate(&startTriCount);
    hipEventCreate(&stopTriCount);

    hipEventCreate(&startNumvertices);
    hipEventCreate(&stopNumvertices);

    hipEventCreate(&startSumTri);
    hipEventCreate(&stopSumTri);

    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfBlocks * threadsPerBlock * sizeof(int));
    hipMalloc(&d_out, 2 * numberOfEdges * sizeof(int));


    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);

    sort(dev_edges,numberOfEdges);


    // Hardcoding the node value 
    hipEventRecord(startNumvertices);
    // numberOfNodes = NumVerticesGPU(numberOfEdges,dev_edges);
    calculateNumVertices(dev_edges, d_out, numberOfEdges * 2);
    hipEventRecord(stopNumvertices);
    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    numberOfNodes = 1 + (*out);
 
    // allocate space for the node array
    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));

    hipEventRecord(startNodeArray1);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2,numberOfNodes);
    hipEventRecord(stopNodeArray1);

    hipDeviceSynchronize();

    // compute the degree of the nodes
    hipEventRecord(startFilter);
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges);
    hipEventRecord(stopFilter);

    hipDeviceSynchronize();

    //remove the filtered edges
    remove(dev_edges,numberOfEdges);

    //get the node array once again
    //note = new size of the edge array is now numberOfEdges

    hipEventRecord(startNodeArray2);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipEventRecord(stopNodeArray2);

    hipDeviceSynchronize(); 

    hipEventRecord(startTriCount);
    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, dev_nodes, result, numberOfEdges);
    hipEventRecord(stopTriCount);

    hipDeviceSynchronize();

    //calculate the number of triangles
    hipEventRecord(startSumTri);
    calculateSum(result, d_out, numberOfBlocks * threadsPerBlock);
    hipEventRecord(stopSumTri);

    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    int numberoftriangles = (*out);

    printf("number of triangles = %d\n",numberoftriangles);

    hipFree(dev_edges);
    hipFree(dev_nodes);
    hipFree(result);
    hipFree(d_out);
    free(out);

    float m1 = 0;
    hipEventElapsedTime(&m1, startNodeArray1, stopNodeArray1);
    printf("CUDA Elapsed Time for Node Array 1 = %f ms\n", m1);

    float m2 = 0;
    hipEventElapsedTime(&m2, startFilter, stopFilter);
    printf("CUDA Elapsed Time for Filter = %f ms\n", m2);

    float m3 = 0;
    hipEventElapsedTime(&m3, startNodeArray2, stopNodeArray2);
    printf("CUDA Elapsed Time for Node Array 2 = %f ms\n", m3);

    float m4 = 0;
    hipEventElapsedTime(&m4, startTriCount, stopTriCount);
    printf("CUDA Elapsed Time for Triangle Counting = %f ms\n", m4);

    float m5 = 0;
    hipEventElapsedTime(&m5, startNumvertices, stopNumvertices);
    printf("CUDA Elapsed Time for num of vertices = %f ms\n", m5);

    float m6 = 0;
    hipEventElapsedTime(&m6, startSumTri, stopSumTri);
    printf("CUDA Elapsed Time for Sume Triangles %f ms\n", m6);

    hipEventDestroy(startNodeArray1);
    hipEventDestroy(stopNodeArray1);

    hipEventDestroy(startNodeArray2);
    hipEventDestroy(stopNodeArray2);

    hipEventDestroy(startFilter);
    hipEventDestroy(stopFilter);

    hipEventDestroy(startTriCount);
    hipEventDestroy(stopTriCount);

    hipEventDestroy(startNumvertices);
    hipEventDestroy(stopNumvertices);

    hipEventDestroy(startSumTri);
    hipEventDestroy(stopSumTri);
    
    double endKernelTime = CycleTimer::currentSeconds();
    double kernelDuration = endKernelTime - startKernelTime;
    printf("KernelDuration: %.3f ms\n", 1000.f * kernelDuration);


}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}
