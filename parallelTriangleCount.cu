#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "lib.h"

using namespace std;

///////////////////////////////////Costants/////////////////////////////////////////////////////////////////////////////
#define threadsPerBlock 1024
#define numberOfBlocks 400
#define FILTER -2
#define inf 0x7f800000 
///////////////////////////////////Costants/////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////DEBUG utility/////////////////////////////////////////////////////////////////////////
// ptr =  cuda device pointer
// size = size of cuda pointer
// msg = custom message
//utility to debug the code
void debug(int *ptr,int size, string msg){

    cout<<msg<<endl;

    int* deb = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb,ptr, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
      cout<<deb[i]<<" ";

    cout<<"\n";

    free(deb);

}

void compare(int *ptr1 , int *ptr2 ,int size){

    int* deb1 = (int*)malloc(size * sizeof(int));
    int* deb2 = (int*)malloc(size * sizeof(int));

    hipMemcpy(deb1,ptr1, size * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(deb2,ptr2, size * sizeof(int), hipMemcpyDeviceToHost);


    for(int i=0 ;i<size;i++)
        if(deb1[i] != deb2[i])
          cout<<i<<" "<<deb1[i]<<" "<<deb2[i]<<endl;

    free(deb1);
    free(deb2);

}


void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}

///////////////////////////////////////////DEBUG utility/////////////////////////////////////////////////////////////////////////



// node array saves the index of the node in edge
// assumption-: array is sorted
// dev_edges = edge array
// dev_nodes = node array
// size = size of the dev_edges
// n = number of nodes
//
__global__ void nodeArray(int* dev_edges, int *dev_nodes,int size, int n){


    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int x,y;

    // bug-: node id that were not present, were not getting updated to zero 
    // eg-: 0 and 1 is not present they should have nodeattay index as 0 [resolved]

    if (idx == 0){

        x = dev_edges[1];
        for(int i=0 ;i<=x;i++)
            dev_nodes[i] = 0;
    }
    // take a step size in grid
    for( int id = idx; id < size/2; id += step){
        
        int edgeIndex = (id * 2) + 1;
        
        x = dev_edges[edgeIndex];
        
        if(id == size/2 - 1)
          y = n;
        
        else  
          y = dev_edges[edgeIndex + 2];
        
        // dealing with missing nodes
        for(int i = x+1 ; i <= y ; i++ ){  
            dev_nodes[i] = id + 1; //always divisble by two
        }

    }
}    

// mark the outgoing edges
// making graph from directed to undirected
// dev_edges = edge array
// dev_nodes = node array
__global__ void filter(int* dev_edges,int* dev_nodes,int numberOfEdges){

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;

    for(id = id; id < numberOfEdges ; id += step){

        int2 sd_pair = ((int2*)dev_edges)[id];

        int sourceDegree = dev_nodes[(sd_pair.x)+1] - dev_nodes[sd_pair.x];
        int destinationDegree = dev_nodes[(sd_pair.y) + 1] - dev_nodes[sd_pair.y]; 

        if(destinationDegree < sourceDegree || (destinationDegree == sourceDegree && sd_pair.y < sd_pair.x)){
            ((int2*)dev_edges)[id] =  make_int2(FILTER, FILTER);

        }         
    }
}   


//////////////////////////////////finding max in the edge array (parallel reduction)////////////////////////////////////////////////////////////
// parallel reduction utitlity
// meant to be operated on a block
__device__ void warp_reduce_max(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] > smem[threadIdx.x] ? 
                        smem[threadIdx.x+512] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+256] > smem[threadIdx.x] ? 
						smem[threadIdx.x+256] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+128] > smem[threadIdx.x] ? 
						smem[threadIdx.x+128] : smem[threadIdx.x]; __syncthreads();

    smem[threadIdx.x] = smem[threadIdx.x+64] > smem[threadIdx.x] ? 
						smem[threadIdx.x+64] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+32] > smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+16] > smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+8] > smem[threadIdx.x] ? 
						smem[threadIdx.x+8] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+4] > smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+2] > smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; __syncthreads();

	smem[threadIdx.x] = smem[threadIdx.x+1] > smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; __syncthreads();
}

// find the max in the edge array
// use to calculate number of nodes
__global__ void find_max(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_max[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;
	int max = -inf;
	int val;
	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
    {
        val = in[idx + i * threadsPerBlock];
        max = val > max ? val : max;

    }

	smem_max[threadIdx.x] = max;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);
	
	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 

}

// meant to be called after every block has found it's maximum
// last step of parallel reduction
__global__ void find_max_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_max[1024];

	int idx = threadIdx.x + remaining;

	int max = -inf;
	int val;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		val = in[idx + iter];
		max = val > max ? val : max;
        iter = i * threadsPerBlock;
    }

	iter = 0;
	for(int i = 1; (iter + threadIdx.x) < num_blocks; i++)
	{
		val = out[threadIdx.x + iter];
		max = val > max ? val : max;
		iter = i * threadsPerBlock;
	}

	smem_max[threadIdx.x] = max;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_max(smem_max);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_max[threadIdx.x]; 
}

//////////////////////////////////finding max in the edge array (parallel reduction)///////////////////////////////////////////////////////////////////////

// cuda wrapper for num of vertices kernel (uses the max kernel)
void calculateNumVertices(int* d_in, int* d_out, int num_elements)
{
		
	int num_blocks = numberOfBlocks;
    int elements_per_block = num_elements / num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;

	find_max<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_max_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);

	
}

// triangle couting kernels
__global__ void trianglecounting(const int* __restrict__ dev_edges,const int* __restrict__ dev_nodes, int* result, int numberOfEdges){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int count  = 0;
 
    // __shared__ int shared[12288];
        //Shared memory optimization (uncomment to use)
    // for (int i=threadIdx.x;i<12288;i=i+1024) {
    //     shared[i] =  dev_nodes[i];
    // }
    // __syncthreads();
    
    for(int iter = idx; iter<numberOfEdges / 2; iter = iter+step){


        int2 se_pair = ((int2*)dev_edges)[iter];

        
        int s_start = dev_nodes[se_pair.x];
        int s_end = dev_nodes[se_pair.x + 1];
        int e_start = dev_nodes[se_pair.y];
        int e_end = dev_nodes[se_pair.y + 1];
        
        int2 s_next,e_next;
        s_next = ((int2*)dev_edges)[s_start];
        e_next = ((int2*)dev_edges)[e_start];

        while(s_start < s_end && e_start < e_end)
        {
            
            int a = s_next.x;
            int b = e_next.x;

            if(a < b) {
                s_start+=1;
                s_next = ((int2*)dev_edges)[s_start];
            }
            else if(a > b) {
                e_start+=1;
                e_next = ((int2*)dev_edges)[e_start];
            }
            else {
                count++;
                s_start+=1;
                s_next = ((int2*)dev_edges)[s_start];
                e_start+=1;
                e_next = ((int2*)dev_edges)[e_start];
            }   
        }
    }
    result[idx] = count;
}

// thrust remove filter
void remove(int* dev_edges,int numberOfEdges) {

    thrust::device_ptr<int> ptr((int*)dev_edges);
    thrust::remove(ptr, ptr + 2*numberOfEdges , FILTER);

}


void sort(int* dev_edges,int numberOfEdges){

    // sort the edges
    thrust::device_ptr<uint64_t> ptr((uint64_t*)dev_edges);
    thrust::sort(ptr, ptr + numberOfEdges);
}

// use this function for comparison purposes
int NumVerticesGPU(int m, int* edges) {
    thrust::device_ptr<int> ptr(edges);
    return 1 + thrust::reduce(ptr, ptr + 2 * m, 0, thrust::maximum<int>());
}

//////////////////////////////////partial sum (parallel reduction)////////////////////////////////////////////////////////////
__device__ void warp_reduce_sum(int smem[1024])
{
    smem[threadIdx.x] = smem[threadIdx.x+512] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+256] + smem[threadIdx.x]; __syncthreads();
    smem[threadIdx.x] = smem[threadIdx.x+128] + smem[threadIdx.x]; __syncthreads();
    smem[threadIdx.x] = smem[threadIdx.x+64] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+32] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+16] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+8] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+4] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+2] + smem[threadIdx.x]; __syncthreads();
	smem[threadIdx.x] = smem[threadIdx.x+1] + smem[threadIdx.x]; __syncthreads();
}

__global__ void find_sum_final(int* in, int* out, int n, int remaining, int num_blocks)
{
	__shared__ int smem_sum[1024];

	int idx = threadIdx.x + remaining;

	int sum = 0;

	// tail part
	int iter = 0;
	for(int i = 1; iter + idx < n; i++)
	{
		sum += in[idx + iter];
        iter = i * threadsPerBlock;
    }
	iter = 0;
	for(int i = 1; (iter + threadIdx.x) < num_blocks; i++)
	{
		sum += out[threadIdx.x + iter];
		iter = i * threadsPerBlock;
	}
	smem_sum[threadIdx.x] = sum;
    __syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_sum(smem_sum);

	if(threadIdx.x == 0)
		out[blockIdx.x] = smem_sum[threadIdx.x]; 
}

__global__ void find_sum(int* in, int* out, int elements_per_block)
{

	__shared__ int smem_sum[1024];

	int idx = threadIdx.x + blockIdx.x * elements_per_block;
	int sum = 0;
	int elements_per_thread = elements_per_block / threadsPerBlock; 
	
    #pragma unroll
    for(int i = 0; i < elements_per_thread; i++)
        sum += in[idx + i * threadsPerBlock];

	smem_sum[threadIdx.x] = sum;
	__syncthreads();

	if(threadIdx.x < 512)
		warp_reduce_sum(smem_sum);

	if(threadIdx.x == 0) 
		out[blockIdx.x] = smem_sum[threadIdx.x]; 
}

void calculateSum(int* d_in, int* d_out, int num_elements)
{
		
	int num_blocks = numberOfBlocks;
    int elements_per_block = num_elements/num_blocks;
	int tail = num_elements - num_blocks * elements_per_block;
	int remaining = num_elements - tail;

	find_sum<<<num_blocks, threadsPerBlock>>>(d_in, d_out, elements_per_block); 
    hipDeviceSynchronize();

	find_sum_final<<<1, threadsPerBlock>>>(d_in, d_out, num_elements, remaining, num_blocks);
}

//////////////////////////////////partial sum (parallel reduction)////////////////////////////////////////////////////////////

// entry point for the cuda kernel
// edges saved as-:
// source1 destination1 source 2 destination 2
void parallelForward(const Edges& edges){


    int numberOfEdges = edges.size();
    int* dev_edges;
    int* dev_nodes;
    int* result;
    int* d_out;
    int numberOfNodes;
    int* out = (int*)malloc(sizeof(int));

    hipEvent_t startNodeArray1, stopNodeArray1, startNodeArray2, stopNodeArray2,startFilter, 
    stopFilter, startTriCount, stopTriCount,startNumvertices,stopNumvertices, startSumTri, stopSumTri, startRemove,stopRemove,startSort,stopSort;

    // timer code
    hipEventCreate(&startNodeArray1);
    hipEventCreate(&stopNodeArray1);

    hipEventCreate(&startNodeArray2);
    hipEventCreate(&stopNodeArray2);

    hipEventCreate(&startFilter);
    hipEventCreate(&stopFilter);

    hipEventCreate(&startTriCount);
    hipEventCreate(&stopTriCount);

    hipEventCreate(&startNumvertices);
    hipEventCreate(&stopNumvertices);

    hipEventCreate(&startSumTri);
    hipEventCreate(&stopSumTri);

    hipEventCreate(&startRemove);
    hipEventCreate(&stopRemove);

    hipEventCreate(&startSort);
    hipEventCreate(&stopSort);

    // transfer data to GPU
    hipMalloc(&dev_edges, 2 * numberOfEdges * sizeof(int));
    hipMalloc(&result, numberOfBlocks * threadsPerBlock * sizeof(int));
    hipMalloc(&d_out, 2 * numberOfEdges * sizeof(int));

    double startKernelTime = CycleTimer::currentSeconds();

    hipMemcpy(dev_edges, edges.data(), numberOfEdges * 2 * sizeof(int),
    hipMemcpyHostToDevice);

    hipEventRecord(startSort);
    sort(dev_edges,numberOfEdges);
    hipEventRecord(stopSort);

    // Hardcoding the node value 
    hipEventRecord(startNumvertices);
    calculateNumVertices(dev_edges, d_out, numberOfEdges * 2);
    hipEventRecord(stopNumvertices);

    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    numberOfNodes = 1 + (*out);
 
    // allocate space for the node array
    hipMalloc(&dev_nodes, (numberOfNodes + 1) * sizeof(int));

    hipEventRecord(startNodeArray1);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges*2,numberOfNodes);
    hipEventRecord(stopNodeArray1);

    hipDeviceSynchronize();

    // compute the degree of the nodes
    hipEventRecord(startFilter);
    filter<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges);
    hipEventRecord(stopFilter);

    hipDeviceSynchronize();

    //remove the filtered edges
    hipEventRecord(startRemove);
    remove(dev_edges,numberOfEdges);
    hipEventRecord(stopRemove);

    //get the node array once again
    //note = new size of the edge array is now numberOfEdges

    hipEventRecord(startNodeArray2);
    nodeArray<<<numberOfBlocks,threadsPerBlock>>>(dev_edges,dev_nodes,numberOfEdges,numberOfNodes);
    hipEventRecord(stopNodeArray2);


    hipDeviceSynchronize(); 

    hipEventRecord(startTriCount);
    trianglecounting<<<numberOfBlocks,threadsPerBlock>>>(dev_edges, dev_nodes, result, numberOfEdges);
    hipEventRecord(stopTriCount);

    hipDeviceSynchronize();

    //calculate the number of triangles
    hipEventRecord(startSumTri);
    calculateSum(result, d_out, numberOfBlocks * threadsPerBlock);
    hipEventRecord(stopSumTri);

    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    int numberoftriangles = (*out);

    printf("number of triangles = %d\n",numberoftriangles);

    double endKernelTime = CycleTimer::currentSeconds();
    double kernelDuration = endKernelTime - startKernelTime;
    printf("KernelDuration: %.3f ms\n", 1000.f * kernelDuration);

    hipFree(dev_edges);
    hipFree(dev_nodes);
    hipFree(result);
    hipFree(d_out);
    free(out);

    float m1 = 0;
    hipEventElapsedTime(&m1, startNumvertices, stopNumvertices);
    printf("CUDA Elapsed Time for num of vertices = %f ms\n", m1);

    float m2 = 0;
    hipEventElapsedTime(&m2, startNodeArray1, stopNodeArray1);
    printf("CUDA Elapsed Time for nodeArray filter 1 = %f ms\n", m2);


    float m3 = 0;
    hipEventElapsedTime(&m3, startFilter, stopFilter);
    printf("CUDA Elapsed Time for edge filter = %f ms\n", m3);

    float m4 = 0;
    hipEventElapsedTime(&m4, startRemove, stopRemove);
    printf("CUDA Elapsed Time for edge Remove %f ms\n", m4);

    float m5 = 0;
    hipEventElapsedTime(&m5, startNodeArray2, stopNodeArray2);
    printf("CUDA Elapsed Time for nodeArray filter 2 = %f ms\n", m5);

    float m6 = 0;
    hipEventElapsedTime(&m6, startTriCount, stopTriCount);
    printf("CUDA Elapsed Time for calculating Triangles %f ms\n", m6);

    float m7 = 0;
    hipEventElapsedTime(&m7, startSumTri, stopSumTri);
    printf("CUDA Elapsed Time for Summing the number of Triangles %f ms\n", m7);

    float m8 = 0;
    hipEventElapsedTime(&m8, startSort, stopSort);
    printf("CUDA Elapsed Time for Sorting %f ms\n", m8);

    hipEventDestroy(startNodeArray1);
    hipEventDestroy(stopNodeArray1);

    hipEventDestroy(startNodeArray2);
    hipEventDestroy(stopNodeArray2);

    hipEventDestroy(startFilter);
    hipEventDestroy(stopFilter);

    hipEventDestroy(startTriCount);
    hipEventDestroy(stopTriCount);

    hipEventDestroy(startNumvertices);
    hipEventDestroy(stopNumvertices);

    hipEventDestroy(startSumTri);
    hipEventDestroy(stopSumTri);

    hipEventDestroy(startRemove);
    hipEventDestroy(stopRemove);

    hipEventDestroy(startSort);
    hipEventDestroy(stopSort);

    // printCudaInfo();
    

}

